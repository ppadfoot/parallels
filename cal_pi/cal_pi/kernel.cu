
#include "hip/hip_runtime.h"

#include <time.h>

/**
 * Using CPU to calculate pi
 * @param a Lower Integral Bounds
 * @param b Upper Integral Bounds
 * @param Integral Value
 */
const int N = 1024 * 1024 * 64;
void pi_by_cpu(double a, double b, double *integral) {
	int i;
	double x, temp = 0;
	for (i = 0; i < N; i++) {
		x = a + (double)(b - a) / N * (i + 0.5);
		temp += 4 / (1 + x * x);
	}
	temp *= (double)(b - a) / N;
	*integral = temp;
}

// Using CUDA device to calculate pi
#include <stdio.h>


#define NBIN N  // Number of bins
#define NUM_BLOCK  64  // Number of thread blocks
#define NUM_THREAD  256  // Number of threads per block	
int tid;
double pi = 0;

// Kernel that executes on the CUDA device
__global__ void cal_pi(double *sum, int nbin, double step, int nthreads, int nblocks) {
	int i;
	double x;
	int idx = blockIdx.x*blockDim.x + threadIdx.x;  // Sequential thread index across the blocks
	for (i = idx; i < nbin; i += nthreads * nblocks) {
		x = (i + 0.5)*step;
		sum[idx] += 4.0 / (1.0 + x * x);
	}
}

// Main routine that executes on the host
int main(void) {

	//Using CPU to calculate pi
	double a, b;
	double   integral;
	clock_t  clockBegin, clockEnd;
	double duration;
	a = 0;
	b = 1;
	clockBegin = clock();
	pi_by_cpu(a, b, &integral);
	clockEnd = clock();
	duration = (double)1000 * (clockEnd - clockBegin) / CLOCKS_PER_SEC;
	printf("CPU Result: %.11lf\n", integral);
	printf("CPU Elapsed time: %.6lfms\n\n", duration);

	// Using CUDA device to calculate pi
	hipEvent_t start, stop;
	hipEventCreate(&start); //event
	hipEventCreate(&stop);  //event
	hipEventRecord(start, 0);  //record time
	float tm;

	dim3 dimGrid(NUM_BLOCK, 1, 1);  // Grid dimensions
	dim3 dimBlock(NUM_THREAD, 1, 1);  // Block dimensions
	double *sumHost, *sumDev;  // Pointer to host & device arrays

	double step = 1.0 / NBIN;  // Step size
	size_t size = NUM_BLOCK * NUM_THREAD * sizeof(double);  //Array memory size
	sumHost = (double *)malloc(size);  //  Allocate array on host
	hipMalloc((void **)&sumDev, size);  // Allocate array on device
	// Initialize array in device to 0
	hipMemset(sumDev, 0, size);
	// Do calculation on device
	cal_pi <<<dimGrid, dimBlock >>> (sumDev, NBIN, step, NUM_THREAD, NUM_BLOCK); // call CUDA kernel
	// Retrieve result from device and store it in host array
	hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
	for (tid = 0; tid < NUM_THREAD*NUM_BLOCK; tid++)
		pi += sumHost[tid];
	pi *= step;

	// Print results
	printf("GPU Result: %.11lf\n", pi);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tm, start, stop);
	printf("GPU Elapsed time:%.6f ms.\n\n", tm);


	// Cleanup
	free(sumHost);
	hipFree(sumDev);
	printf("Press to exit.\n");
	getchar();
	return 0;
}

